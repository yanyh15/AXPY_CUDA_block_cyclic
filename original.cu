#include <iostream>
#include <hip/hip_runtime.h>

template<typename T>
__global__ void axpy(T a, T *x, T *y) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  y[i] += a * x[i];
}


int main(int argc, char* argv[]) {
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};

  float* device_x;
  float* device_y;

  hipMalloc(&device_x, kDataLen * sizeof(float));

  hipMalloc(&device_y, kDataLen * sizeof(float));

  hipMalloc(&device_y, kDataLen * sizeof(double));

  hipMemcpy(device_x, host_x, kDataLen * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_y, host_y, kDataLen * sizeof(float), hipMemcpyHostToDevice);

  axpy<<<1, kDataLen>>>(a, device_x, device_y);

  hipDeviceSynchronize();

  hipMemcpy(host_y, device_y, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << "\n";
  }

  hipDeviceReset();
  return 0;
}

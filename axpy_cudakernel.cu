#include "hip/hip_runtime.h"
#include "axpy.h"

__global__ 
void
axpy_cudakernel_1perThread(REAL* x, REAL* y, int n, REAL a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) y[i] += a*x[i];
}

/* block distribution of loop iteration */
__global__ 
void axpy_cudakernel_block(REAL* x, REAL* y, int n, REAL a) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = gridDim.x * blockDim.x;

	int block_size = n / total_threads; //dividable, TODO handle non-dividiable later
	
	int start_index = thread_num * block_size;
	int stop_index = start_index + block_size;
	int i;
        for (i=start_index; i<stop_index; i++) {
		if (i < n) y[i] += a*x[i];
	}
}

/* cyclic distribution of loop distribution */
__global__
void axpy_cudakernel_cyclic(REAL* x, REAL* y, int n, REAL a) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = gridDim.x * blockDim.x;
	
	int i;
	for (i=thread_num; i<n; i+=total_threads) { 
		if (i < n) y[i] += a*x[i];
	}
}

/* block_cyclic distribution of loop distribution */
__global__
void axpy_cudakernel_cyclic(REAL* x, REAL* y, int n, REAL a, int block_size) {
	int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
	int total_threads = gridDim.x * blockDim.x;
	
	int i;
	for (i=thread_num; i<n; i+=total_threads) { 
		if (i < n) y[i] += a*x[i];
	}
}

void axpy_cuda(REAL* x, REAL* y, int n, REAL a) {
  REAL *d_x, *d_y;
  hipMalloc(&d_x, n*sizeof(REAL));
  hipMalloc(&d_y, n*sizeof(REAL));

  hipMemcpy(d_x, x, n*sizeof(REAL), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, n*sizeof(REAL), hipMemcpyHostToDevice);

  // Perform axpy elements
  axpy_cudakernel_1perThread<<<(n+255)/256, 256>>>(d_x, d_y, n, a);
  axpy_cudakernel_block<<<1024, 256>>>(d_x, d_y, n, a);
  axpy_cudakernel_cyclic<<<1024, 256>>>(d_x, d_y, n, a);

  hipMemcpy(y, d_y, n*sizeof(REAL), hipMemcpyDeviceToHost);
  hipFree(d_x);
  hipFree(d_y);
}

#include <iostream>
#include <hip/hip_runtime.h>

template<typename T>
__global__ void axpy(T a, T *x, T *y) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  y[i] += a * x[i];
}
// cyclic distribution ⇒ coalesced memory access. In OpenMP,  If you use schedule(static:chunk:1), it forces cyclic distribution. (dynamic:chunk:1) is not cyclic, but could end up to be cyclic
__global__ void axpy_cudakernel_cyclic(float *y, float *x, float a, int kDataLen) {
   // int i;
    int thread_num = threadIdx.x + blockIdx.x * blockDim.x;
    int total_threads = gridDim.x * blockDim.x;
    //int block_size = kDataLen / total_threads; //dividable, TODO handle non-dividiable later
    if (thread_num < kDataLen) {
        int start_index = thread_num;
        for (int i=start_index; i<kDataLen; i+=total_threads) 
            y[i] += a*x[i];
    }
}



int main(int argc, char* argv[]) {
  const int kDataLen = 10240;

  float a = 2.0f;
  float host_x[kDataLen];
  float host_y[kDataLen];
  for(int i=0; i<10240; i++) host_x[i] = i;//rand();
  for(int i=0; i<10240; i++) host_y[i] = i;//rand();
  //float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
 /// float host_y[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};

  float* device_x;
  float* device_y;

  hipMalloc(&device_x, kDataLen * sizeof(float));

  hipMalloc(&device_y, kDataLen * sizeof(float));

  hipMalloc(&device_y, kDataLen * sizeof(double));

  hipMemcpy(device_x, host_x, kDataLen * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(device_y, host_y, kDataLen * sizeof(float), hipMemcpyHostToDevice);

  //dim3 block_size(16, 16);
  //dim3 grid_size(2, 2);
  for(int i = 0; i<100; i++){
    axpy_cudakernel_cyclic<<<2, 256>>>( device_y, device_x, a, kDataLen);
  }

  hipDeviceSynchronize();

  hipMemcpy(host_y, device_y, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

  // Print the results.
  /*for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << "\n";
  }*/

  hipDeviceReset();
  return 0;
}
